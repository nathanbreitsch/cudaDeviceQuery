
#include <hip/hip_runtime.h>
#include <stdio.h>


//cudaGetDeviceCount(addr): assigns count to addr, returns status value cudaSuccess, cudaErrorNoDevice, cudaErrorInsufficientDriver 
//cudaDeviceProp: struct for device props
//cudaGetDeviceProperties(addr, index): assigns device properties to addr, returns status value cudaSuccess, cudaErrorInvalidDevice



int main(){
	int device_count;
	hipError_t device_count_status = hipGetDeviceCount(&device_count);
	if(device_count_status == hipErrorNoDevice){
		printf("No devices were found \n");
		return 0;
	}
	else if(device_count_status == hipErrorInsufficientDriver){
		printf("Insufficient Driver \n");
		return 0;
	}
	hipDeviceProp_t current_device_properties;
	for(int i = 0; i < device_count; ++i){
		hipError_t current_device_status = hipGetDeviceProperties(&current_device_properties, i);
		if(current_device_status == hipErrorInvalidDevice){
			printf("Device number:%d is invalid \n", i);
		}
		else{
			char* name = current_device_properties.name;
			int multiprocessor_count = current_device_properties.multiProcessorCount;
			size_t shared_mem_per_block = current_device_properties.sharedMemPerBlock;
			size_t total_global_memory = current_device_properties.totalGlobalMem;
			int max_threads_per_block = current_device_properties.maxThreadsPerBlock;
			int* block_dim_limits = current_device_properties.maxThreadsDim;
			int* grid_dim_limits = current_device_properties.maxGridSize;
			printf("Stats for device #%d:\n", i);
			printf("name: ");
			printf(name);
			printf("\n");
			printf("number of multiprocessors: %d\n", multiprocessor_count);
			printf("shared memery per block: %d\n", shared_mem_per_block);
			printf("total global memory: %d\n", total_global_memory);
			printf("max threads per block: %d\n", max_threads_per_block);
			printf("maximum block dimensions: %d X %d X %d\n", block_dim_limits[0], block_dim_limits[1], block_dim_limits[2]);
			printf("maximum grid dimensions: %d X %d X %d\n", grid_dim_limits[0], grid_dim_limits[1], grid_dim_limits[2]);
			
		}
	}

	return 0;
}
