
#include <hip/hip_runtime.h>
#include <stdio.h>


//cudaGetDeviceCount(addr): assigns count to addr, returns status value cudaSuccess, cudaErrorNoDevice, cudaErrorInsufficientDriver 
//cudaDeviceProp: struct for device props
//cudaGetDeviceProperties(addr, index): assigns device properties to addr, returns status value cudaSuccess, cudaErrorInvalidDevice



int main(){
	int device_count;
	hipError_t device_count_status = hipGetDeviceCount(&device_count);
	if(device_count_status == hipErrorNoDevice){
		printf("No devices were found");
		return 0;
	}
	else if(device_count_status == hipErrorInsufficientDriver){
		printf("Insufficient Driver");
		return 0;
	}
	hipDeviceProp_t current_device_properties;
	for(int i = 0; i < device_count; ++i){
		hipError_t current_device_status = hipGetDeviceProperties(&current_device_properties, i);
		if(current_device_status == hipErrorInvalidDevice){
			printf("Device number:%d is invalid", i);
		}
		else{
			printf("time to list the stuff");
		}
	}

	return 0;
}
